#include "hip/hip_runtime.h"
#include "Utils.h"
#include <hip/hip_math_constants.h>


__global__ void computeWeierstrassFilterCuda (double *f, double *s, double sigma, 
	int *isize, int *istart, int *n) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int k = threadIdx.z + blockDim.z * blockIdx.z;

	double X, Y, Z, Xp, Yp, Zp, twopi;
	int hx, hy, hz;
	int64_t ptr;
	twopi = 2. * HIP_PI;
	hx = twopi / n[0];
	hy = twopi / n[1];
	hz = twopi / n[2];

	X = (istart[0] + i) * hx;
	Xp = X - twopi;
	Y = (istart[1] + j) * hy;
	Yp = Y - twopi;
	Z = (istart[2] + k) * hz;
	Zp = Z - twopi;
	ptr = i * isize[1] * isize[2] + j * isize[2] + k;
	f[ptr] = exp((-X * X - Y * Y - Z * Z) / sigma / sigma / 2.0)
			+ exp((-Xp * Xp - Yp * Yp - Zp * Zp) / sigma / sigma / 2.0);

	f[ptr] += exp((-Xp * Xp - Y * Y - Z * Z) / sigma / sigma / 2.0)
			+ exp((-X * X - Yp * Yp - Z * Z) / sigma / sigma / 2.0);

	f[ptr] += exp((-X * X - Y * Y - Zp * Zp) / sigma / sigma / 2.0)
			+ exp((-Xp * Xp - Yp * Yp - Z * Z) / sigma / sigma / 2.0);

	f[ptr] += exp((-Xp * Xp - Y * Y - Zp * Zp) / sigma / sigma / 2.0)
			+ exp((-X * X - Yp * Yp - Zp * Zp) / sigma / sigma / 2.0);

	if (f[ptr] != f[ptr])
		f[ptr] = 0.; // To avoid Nan
	s += f[ptr];
}

__global__ void hadamardComplexProductCuda (std::complex<double> *y, std::complex<double> *x, double *alph) {
	int i = threadIdx.x;
	y[i] *= (x[i] * (*alph));
}