#include "hip/hip_runtime.h"
#include "UtilsCuda.h"
//#include "Common.cuh"

__constant__ int isize_cuda[3], istart_cuda[3], osize_cuda[3], ostart_cuda[3], n_cuda[3];

void initCudaConstants (int *isize, int *osize, int *istart, int *ostart, int *n) {
	hipMemcpyToSymbol(HIP_SYMBOL(isize_cuda), isize, 3 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(osize_cuda), osize, 3 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(istart_cuda), istart, 3 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(ostart_cuda), ostart, 3 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(n_cuda), n, 3 * sizeof(int));
}

__global__ void hadamardComplexProduct (CudaComplexType *y, ScalarType *x) {
	int64_t i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < osize_cuda[0] * osize_cuda[1] * osize_cuda[2]) 
		y[i] = cuComplexMultiply (y[i], makeCudaComplexType(x[i], 0.));
}

__global__ void hadamardComplexProduct (CudaComplexType *y, CudaComplexType *x) {
	int64_t i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < osize_cuda[0] * osize_cuda[1] * osize_cuda[2]) 
		y[i] = cuComplexMultiply (y[i], x[i]);
}

__global__ void computeMagnitude (ScalarType *mag_ptr, ScalarType *x_ptr, ScalarType *y_ptr, ScalarType *z_ptr, int64_t sz) {
	int64_t i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < sz)  
		mag_ptr[i] = sqrt (x_ptr[i] * x_ptr[i] + y_ptr[i] * y_ptr[i] + z_ptr[i] * z_ptr[i]);
}

__global__ void setCoords (ScalarType *x_ptr, ScalarType *y_ptr, ScalarType *z_ptr) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int k = threadIdx.z + blockDim.z * blockIdx.z;

	int64_t ptr = i * isize_cuda[1] * isize_cuda[2] + j * isize_cuda[2] + k;

	if (ptr < isize_cuda[0] * isize_cuda[1] * isize_cuda[2]) {
		// ScalarType hx, hy, hz;
		// ScalarType twopi = 2. * HIP_PI;
		// hx = twopi / n_cuda[0];
		// hy = twopi / n_cuda[1];
		// hz = twopi / n_cuda[2];

		x_ptr[ptr] = static_cast<ScalarType> (i + istart_cuda[0]);
        y_ptr[ptr] = static_cast<ScalarType> (j + istart_cuda[1]);
        z_ptr[ptr] = static_cast<ScalarType> (k + istart_cuda[2]);    
    }
}

__global__ void clipVector (ScalarType *x_ptr) {
	int64_t i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < isize_cuda[0] * isize_cuda[1] * isize_cuda[2]) {
		x_ptr[i] = (x_ptr[i] <= 0.) ? 0. : x_ptr[i];
	}
}

__global__ void clipVectorAbove (ScalarType *x_ptr) {
	int64_t i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < isize_cuda[0] * isize_cuda[1] * isize_cuda[2]) {
		x_ptr[i] = (x_ptr[i] > 1.) ? 1. : x_ptr[i];
	}
}

void setCoordsCuda (ScalarType *x_ptr, ScalarType *y_ptr, ScalarType *z_ptr, int *sz) {
	int n_th_x = N_THREADS_X;
	int n_th_y = N_THREADS_Y;
	int n_th_z = N_THREADS_Z;
	dim3 n_threads (n_th_x, n_th_y, n_th_z);
	dim3 n_blocks ((sz[0] + n_th_x - 1) / n_th_x, (sz[1] + n_th_y - 1) / n_th_y, (sz[2] + n_th_z - 1) / n_th_z);

	setCoords <<< n_blocks, n_threads >>> (x_ptr, y_ptr, z_ptr);

	hipDeviceSynchronize();
	cudaCheckKernelError ();
}

void computeMagnitudeCuda (ScalarType *mag_ptr, ScalarType *x_ptr, ScalarType *y_ptr, ScalarType *z_ptr, int64_t sz) {
	int n_th = N_THREADS;

	computeMagnitude <<< (sz + n_th - 1) / n_th, n_th >>> (mag_ptr, x_ptr, y_ptr, z_ptr, sz);

	hipDeviceSynchronize();
	cudaCheckKernelError ();
}

void hadamardComplexProductCuda (CudaComplexType *y, ScalarType *x, int *sz) {
	int n_th = N_THREADS;

	hadamardComplexProduct <<< ((sz[0] * sz[1] * sz[2]) + n_th - 1)/ n_th, n_th >>> (y, x);

	hipDeviceSynchronize();
	cudaCheckKernelError ();
}

void hadamardComplexProductCuda (CudaComplexType *y, CudaComplexType *x, int *sz) {
	try	{
		thrust::device_ptr<thrust::complex<ScalarType>> y_thrust, x_thrust;
	    y_thrust = thrust::device_pointer_cast ((thrust::complex<ScalarType>*)y);
	    x_thrust = thrust::device_pointer_cast ((thrust::complex<ScalarType>*)x);

	    thrust::transform(y_thrust, y_thrust + (sz[0] * sz[1] * sz[2]), x_thrust, y_thrust, thrust::multiplies<thrust::complex<ScalarType>>());
	} catch (thrust::system_error &e) {
		std::cerr << "Thrust reduce error: " << e.what() << std::endl;
	}

	hipDeviceSynchronize();
}

void clipVectorCuda (ScalarType *x_ptr, int64_t sz) {
	int n_th = N_THREADS;

	clipVector <<< (sz + n_th - 1) / n_th, n_th >>> (x_ptr);

	hipDeviceSynchronize();
	cudaCheckKernelError ();	
}

void clipVectorAboveCuda (ScalarType *x_ptr, int64_t sz) {
	int n_th = N_THREADS;

	clipVectorAbove <<< (sz + n_th - 1) / n_th, n_th >>> (x_ptr);

	hipDeviceSynchronize();
	cudaCheckKernelError ();	
}

void vecMaxCuda (ScalarType *x, int *loc, ScalarType *val, int sz) {
	// use thrust for vec max
	try {
		thrust::device_ptr<ScalarType> x_thrust;
		x_thrust = thrust::device_pointer_cast (x);
		// find the max itr
		thrust::device_vector<ScalarType>::iterator it = thrust::max_element(x_thrust, x_thrust + sz);
		// find the position
		thrust::device_ptr<ScalarType> max_pos = thrust::device_pointer_cast(&it[0]);
		if (loc != NULL)
			*loc = max_pos - x_thrust;
		*val = *it;
	} catch (thrust::system_error &e) {
		std::cerr << "Thrust vector maximum error: " << e.what() << std::endl;
	}

	hipDeviceSynchronize();
}

void vecScatterCuda(ScalarType *f, ScalarType *f_scatter, ScalarType *seq, int64_t sz) {
	try {
		thrust::device_ptr<ScalarType> f_thrust;
		f_thrust = thrust::device_pointer_cast (f);
		thrust::device_ptr<ScalarType> f_scat_thrust;
		f_scat_thrust = thrust::device_pointer_cast (f_scatter);
    thrust::device_ptr<ScalarType> seq_thrust;
    seq_thrust = thrust::device_pointer_cast (seq);
		thrust::scatter(f_thrust, f_thrust + sz, seq_thrust, f_scat_thrust);
	} catch (thrust::system_error &e) {
		std::cerr << "Thrust scatter error: " << e.what() << std::endl;
	}

	hipDeviceSynchronize();

}

void vecSortCuda(ScalarType *f, int64_t sz) {
	// use thrust for sort
	try {
		thrust::device_ptr<ScalarType> f_thrust;
		f_thrust = thrust::device_pointer_cast (f);
//		if (seq == NULL) {
		thrust::sort (f_thrust, f_thrust + sz);
//		} else {
//      thrust::device_ptr<ScalarType> seq_thrust;
//      seq_thrust = thrust::device_pointer_cast (seq);
//		  thrust::sort_by_key(f_thrust, f_thrust + sz, seq_thrust);
//    }
	} catch (thrust::system_error &e) {
		std::cerr << "Thrust sorting error: " << e.what() << std::endl;
	}

	hipDeviceSynchronize();
}

void setSequenceCuda(ScalarType *f, int64_t sz) {
	// use thrust for to set sequence
	try {
		thrust::device_ptr<ScalarType> f_thrust;
		f_thrust = thrust::device_pointer_cast (f);
		thrust::sequence(f_thrust, f_thrust + sz);
	} catch (thrust::system_error &e) {
		std::cerr << "Thrust sequence set error: " << e.what() << std::endl;
	}

	hipDeviceSynchronize();
}

void vecSumCuda(ScalarType *f, ScalarType *sum, int64_t sz) {
	// use thrust for reduction
	try {
		thrust::device_ptr<ScalarType> f_thrust;
		f_thrust = thrust::device_pointer_cast (f);
		(*sum) = thrust::reduce (f_thrust, f_thrust + sz);
	} catch (thrust::system_error &e) {
		std::cerr << "Thrust reduce error: " << e.what() << std::endl;
	}

	hipDeviceSynchronize();
}

__global__ void copyDoubleToFloat(float *dst, double *src, int64_t sz) {
    int64_t i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < sz) {
        dst[i] = (float)src[i];
    }
}

void copyDoubleToFloatCuda (float *dst, double *src, int64_t sz) {
    int n_th = N_THREADS;

    copyDoubleToFloat <<<  (sz + n_th - 1) / n_th, n_th >>> (dst, src, sz);

    hipDeviceSynchronize();
    cudaCheckKernelError();
}

__global__ void copyFloatToDouble(double *dst, float *src, int64_t sz) {
        int64_t i = threadIdx.x + blockDim.x * blockIdx.x;

        if (i < sz) {
            dst[i] = (double)src[i];
        }
}

__global__ void computeIndicatorFunction(ScalarType *i_ptr, ScalarType *x_ptr, ScalarType x_star, ScalarType threshold, int64_t sz) {

  int64_t i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < sz) {
    if (abs(x_ptr[i] - x_star) < threshold) {
      i_ptr[i] = 1;
    } else {
      i_ptr[i] = 0;
    }
  }
}
  
void copyFloatToDoubleCuda (double *dst, float *src, int64_t sz) {
    int n_th = N_THREADS;

    copyFloatToDouble <<<  (sz + n_th - 1) / n_th, n_th >>> (dst, src, sz);

    hipDeviceSynchronize();
    cudaCheckKernelError();
}

void computeIndicatorFunctionCuda(ScalarType *i_ptr, ScalarType *x_ptr, ScalarType x_star, ScalarType threshold, int64_t sz) {
    int n_th = N_THREADS;

    computeIndicatorFunction <<<  (sz + n_th - 1) / n_th, n_th >>> (i_ptr, x_ptr, x_star, threshold, sz);

    hipDeviceSynchronize();
    cudaCheckKernelError();
}

__global__ void smoothHeavisideFunction(ScalarType *x_ptr, ScalarType *y_ptr, ScalarType shapeFactor int64_t sz) {
  
  int64_t i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < sz) {
    y_ptr[i] = 1 / (1 + std::exp(- shapeFactor * x_ptr[i]))
  }
  
}

void smoothHeavisideFunctionCuda(ScalarType *x_ptr, ScalarType *y_ptr, ScalarType shapeFactor, int64_t sz) {
  int n_th = N_THREADS; 
  
  smoothHeavisideFunction <<< (sz + n_th - 1) / n_th, n_th >>> (x_ptr, y_ptr, shapeFactor, sz); 

  hipDeviceSynchronize(); 
  cudaCheckKernelError();
  
}

__global__ void multispeciesObsOperator (ScalarType *p_ptr, ScalarType *n_ptr, ScalarType *i_ptr, ScalarType *w_ptr, ScalarType *g_ptr, ScalarType *f_ptr, ScalarType *Oc_ptr, ScalarType *Op_ptr, ScalarType *On_ptr, ScalarType *Ol_ptr, ScalarType shapeFactorEdema, ScalarType shapeFactor, ScalarType thresEdema, int64_t sz} {

  int64_t i = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < sz) {
    ScalarType c = p_ptr[i] + n_ptr[i] + i_ptr[i];
    
    Oc_ptr[i] = (1/(1+std::exp(-shapeFactor * (c - w_ptr[i])))) * 
                (1/(1+std::exp(-shapeFactor * (c - g_ptr[i])))) * 
                (1/(1+std::exp(-shapeFactor * (c - f_ptr[i]))));
    
    Op_ptr[i] = (1/(1+std::exp(-shapeFactor * (p_ptr[i] - n_ptr[i])))) * 
                (1/(1+std::exp(-shapeFactor * (p_ptr[i] - i_ptr[i])))) * 
                Oc_ptr[i];
    
    On_ptr[i] = (1/(1+std::exp(-shapeFactor * (n_ptr[i] - p_ptr[i])))) *
                (1/(1+std::exp(-shapeFactor * (n_ptr[i] - i_ptr[i])))) *
                Oc_ptr[i];
    
    Ol_ptr[i] = (1 - Op_ptr[i] - On_ptr[i]) * (1/(1 + std::exp(-shapeFactorEdema * (i_ptr[i] - thresEdema))))

  } 
}


void multispeciesObsOperatorsCuda (ScalarType *p_ptr, ScalarType *n_ptr, ScalarType *i_ptr, ScalarType *w_ptr, ScalarType *g_ptr, ScalarType *f_ptr, ScalarType *Oc_ptr, ScalarType *Op_ptr, ScalarType *On_ptr, ScalarType *Ol_ptr, ScalarType shapeFactorEdema, ScalarType shapeFactor, ScalarType thresEdema, int64_t sz) {

  int n_th = N_THREADS;
  
  multispeciesObsOperators <<< (sz + n_th - 1) / n_th, n_th >>> (p_ptr, n_ptr, i_ptr, w_ptr, g_ptr, f_ptr, Oc_ptr, Op_ptr, On_ptr, Ol_ptr, shapeFactorEdema, shapeFactor, thresEdema, sz); 

  hipDeviceSynchronize();
  cudaCheckKernelError();

}


