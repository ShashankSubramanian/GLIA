#include "hip/hip_runtime.h"
#include "ElasticitySolver.h"


__constant__ int n_cuda[3], ostart_cuda[3], osize_cuda[3];

void initElasticityCudaConstants(int *n, int *ostart, int *osize) {
	hipMemcpyToSymbol(HIP_SYMBOL(ostart_cuda), ostart, 3 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(n_cuda), n, 3 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(osize_cuda), osize, 3 * sizeof(int));
}


__global__ void computeScreening (ScalarType *screen_ptr, ScalarType *c_ptr, ScalarType *bg_ptr, ScalarType screen_low, ScalarType screen_high, int64_t sz) {
	int64_t i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < sz) {
		ScalarType c_threshold = 0.005;
		screen_ptr[i] = (c_ptr[i] >= c_threshold) ? screen_low : screen_high;
		if (bg_ptr[i] > 0.95) screen_ptr[i] = 1E6; // screen out the background completely to ensure no movement
	}
}

__global__ void computeTumorLame (ScalarType *mu_ptr, ScalarType *lam_ptr, ScalarType *c_ptr, ScalarType mu_tumor, ScalarType lam_tumor, int64_t sz) {
	int64_t i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < sz) {
		// positivity clipping for mu, lam because tissues are no longer clipped
		mu_ptr[i] = (mu_ptr[i] > 0) ? mu_ptr[i] : 0;
        lam_ptr[i] = (lam_ptr[i] > 0) ? lam_ptr[i] : 0;

		mu_ptr[i] += (c_ptr[i] > 0) ? (mu_tumor * c_ptr[i]) : 0;
        lam_ptr[i] += (c_ptr[i] > 0) ? (lam_tumor * c_ptr[i]) : 0;
	}
}

__global__ void precFactorElasticity (CudaComplexType *ux_hat, CudaComplexType *uy_hat, CudaComplexType *uz_hat, CudaComplexType *fx_hat, CudaComplexType *fy_hat, CudaComplexType *fz_hat, ScalarType lam_avg, ScalarType mu_avg, ScalarType screen_avg) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int k = threadIdx.z + blockDim.z * blockIdx.z;

	int64_t ptr = i * osize_cuda[1] * osize_cuda[2] + j * osize_cuda[2] + k;

	if (i < osize_cuda[0] && j < osize_cuda[1] && k < osize_cuda[2]) {
		ScalarType s1, s2, s1_square, s3, scale;
	    int64_t wx, wy, wz;
	    ScalarType wTw, wTf_real, wTf_imag;
	    int64_t x_global, y_global, z_global;

	    ScalarType factor = 1.0 / (n_cuda[0] * n_cuda[1] * n_cuda[2]);
	    s2 = lam_avg + mu_avg;


		x_global = i + ostart_cuda[0];
		y_global = j + ostart_cuda[1];
		z_global = k + ostart_cuda[2];

		wx = x_global;
		if (x_global > n_cuda[0] / 2) // symmetric frequencies
			wx -= n_cuda[0];
		if (x_global == n_cuda[0] / 2) // nyquist frequency
			wx = 0;

		wy = y_global;
		if (y_global > n_cuda[1] / 2) // symmetric frequencies
			wy -= n_cuda[1];
		if (y_global == n_cuda[1] / 2) // nyquist frequency
			wy = 0;

		wz = z_global;
		if (z_global > n_cuda[2] / 2) // symmetric frequencies
			wz -= n_cuda[2];
		if (z_global == n_cuda[2] / 2) // nyquist frequency
			wz = 0;

		wTw = -1.0 * (wx * wx + wy * wy + wz * wz);

		s1 = -screen_avg + mu_avg * wTw;
		s1_square = s1 * s1;
		s3 = 1.0 / (1.0 + (wTw * s2) / s1);

		wTf_real = wx * fx_hat[ptr].x + wy * fy_hat[ptr].x + wz * fz_hat[ptr].x;
		wTf_imag = wx * fx_hat[ptr].y + wy * fy_hat[ptr].y + wz * fz_hat[ptr].y;

		// real part
		scale = -1.0 * wx * wTf_real;
		ux_hat[ptr].x = factor * (fx_hat[ptr].x * (1.0 / s1) - (1.0 / s1_square) * s2 * s3 * scale); 
		// imaginary part
		scale = -1.0 * wx * wTf_imag;
		ux_hat[ptr].y = factor * (fx_hat[ptr].y * (1.0 / s1) - (1.0 / s1_square) * s2 * s3 * scale); 

		// real part
		scale = -1.0 * wy * wTf_real;
		uy_hat[ptr].x = factor * (fy_hat[ptr].x * (1.0 / s1) - (1.0 / s1_square) * s2 * s3 * scale); 
		// imaginary part
		scale = -1.0 * wy * wTf_imag;
		uy_hat[ptr].y = factor * (fy_hat[ptr].y * (1.0 / s1) - (1.0 / s1_square) * s2 * s3 * scale); 

		// real part
		scale = -1.0 * wz * wTf_real;
		uz_hat[ptr].x = factor * (fz_hat[ptr].x * (1.0 / s1) - (1.0 / s1_square) * s2 * s3 * scale); 
		// imaginary part
		scale = -1.0 * wz * wTf_imag;
		uz_hat[ptr].y = factor * (fz_hat[ptr].y * (1.0 / s1) - (1.0 / s1_square) * s2 * s3 * scale); 
	}
}

// computes the jacobian, stress tensors
__global__ void computeStressQuants(ScalarType **gradu_ptr, ScalarType *jac_ptr, ScalarType *trace_ptr, ScalarType *max_shear_ptr, ScalarType *mu_ptr, ScalarType *lam_ptr, int64_t sz) {
	int64_t i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < sz) {
    ScalarType F0,F1,F2,F3,F4,F5,F6,F7,F8;
    ScalarType S0,S1,S2,S3,S4,S5,S6,S7,S8;
    ScalarType trE;

    F0 = gradu_ptr[0][i] + 1;
    F1 = gradu_ptr[1][i];
    F2 = gradu_ptr[2][i];
    F3 = gradu_ptr[3][i];
    F4 = gradu_ptr[4][i] + 1;
    F5 = gradu_ptr[5][i];
    F6 = gradu_ptr[6][i];
    F7 = gradu_ptr[7][i];
    F8 = gradu_ptr[8][i] + 1;

    // determinant of F
    jac_ptr[i] = F0 * (F4*F8 - F7*F5) - F1 * (F3*F8 - F6*F5) + F2 * (F3*F7 - F6*F4);

    // E = 0.5* (FTF - I); or 0.5 * (FT + F) - I if high order terms are dropped
    S0 = 0.5 * (F0 + F0) - 1; 
    S1 = 0.5 * (F3 + F1);
    S2 = 0.5 * (F6 + F2);
    S3 = 0.5 * (F1 + F3);
    S4 = 0.5 * (F4 + F4) - 1;
    S5 = 0.5 * (F7 + F5);
    S6 = 0.5 * (F2 + F6);
    S7 = 0.5 * (F5 + F7);
    S8 = 0.5 * (F8 + F8) - 1;
   
    trE = S0 + S4 + S8;
    // S = lam tr(E) I + 2 mu E
    S0 = lam_ptr[i] * trE + 2 * mu_ptr[i] * S0;
    S1 = 2 * mu_ptr[i] * S1;
    S2 = 2 * mu_ptr[i] * S2;
    S3 = 2 * mu_ptr[i] * S3;
    S4 = lam_ptr[i] * trE + 2 * mu_ptr[i] * S4;
    S5 = 2 * mu_ptr[i] * S5;
    S6 = 2 * mu_ptr[i] * S6;
    S7 = 2 * mu_ptr[i] * S7;
    S8 = lam_ptr[i] * trE + 2 * mu_ptr[i] * S8;
    
    trace_ptr[i] = S0 + S4 + S8; // trace of stress tensor

    // compute eigenvalues
    //Given a real symmetric 3x3 matrix A, compute the eigenvalues
    //Note that acos and cos operate on angles in radians

    // Reuse F, no loops
    // F6, F7, F8 are eigenvalues

    F0 = S1*S1 + S2*S2 + S5*S5; // upper triangle

    if (F0 == 0) {
      // A is diagonal.
      F6 = S0;
      F7 = S4;
      F8 = S8;
    } else {
      F1 = trace_ptr[i]/3;
      F2 = (S0 - F1)*(S0 - F1) + (S4 - F1)*(S4 - F1) + (S8 - F1)*(S8 - F1) + 2*F0;
      F2 = std::sqrt(F2 / 6);
      F3 = (1 / F2);
      // B = (1 / p) * (A - q * I; I is the identity matrix
      S0 = (S0 - F1);
      S1 = S1;
      S2 = S2;
      S3 = S3;
      S4 = (S4 - F1);
      S5 = S5;
      S6 = S6;
      S7 = S7;
      S8 = (S8 - F1);
      // r = det(B)/2
      F4 = F3 * F3 * F3 * 0.5 * (S0 * (S4*S8 - S7*S5) - S1 * (S3*S8 - S6*S5) + S2 * (S3*S7 - S6*S4));

      // In exact arithmetic for a symmetric matrix  -1 <= r <= 1
      // but computation error can leave it slightly outside this range.
      if (F4 <= -1) { 
        F5 = M_PI / 3;
      } else if (F4 >= 1) {
        F5 = 0;
      } else {
        F5 = acos(F4) / 3;
      }

      // the eigenvalues satisfy eig3 <= eig2 <= eig1
      F6 = F1 + 2 * F2 * cos(F5);
      F7 = F1 + 2 * F2 * cos(F5 + (2*HIP_PI/3));
      F8 = 3 * F1 - F6 - F7;     //since trace(A) = eig1 + eig2 + eig3
    }

    // insertion sort eigenvalues
    // reuse trE as temp, no loops
    if (F7 < F6) {
      // swap
      trE = F6;
      F6 = F7;
      F7 = trE;
    }
    if (F8 < F7) {
      trE = F7;
      F7 = F8;
      F8 = trE;

      if (F7 < F6) {
        trE = F6;
        F6 = F7;
        F7 = trE;
      }
    }

    // max_shear
    max_shear_ptr[i] = 0.5 * (F8 - F6);
  }
}

void computeScreeningCuda (ScalarType *screen_ptr, ScalarType *c_ptr, ScalarType *bg_ptr, ScalarType screen_low, ScalarType screen_high, int64_t sz) {
	int n_th = N_THREADS;

	computeScreening <<< (sz + n_th - 1) / n_th, n_th >>> (screen_ptr, c_ptr, bg_ptr, screen_low, screen_high, sz);

	hipDeviceSynchronize();
	cudaCheckKernelError ();
}

void computeTumorLameCuda (ScalarType *mu_ptr, ScalarType *lam_ptr, ScalarType *c_ptr, ScalarType mu_tumor, ScalarType lam_tumor, int64_t sz) {
	int n_th = N_THREADS;

	computeTumorLame <<< (sz + n_th - 1) / n_th, n_th >>> (mu_ptr, lam_ptr, c_ptr, mu_tumor, lam_tumor, sz);

	hipDeviceSynchronize();
	cudaCheckKernelError ();
}

void precFactorElasticityCuda (CudaComplexType *ux_hat, CudaComplexType *uy_hat, CudaComplexType *uz_hat, CudaComplexType *fx_hat, CudaComplexType *fy_hat, CudaComplexType *fz_hat, ScalarType lam_avg, ScalarType mu_avg, ScalarType screen_avg, int *sz) {
	int n_th_x = N_THREADS_X;
	int n_th_y = N_THREADS_Y;
	int n_th_z = N_THREADS_Z;
	dim3 n_threads (n_th_x, n_th_y, n_th_z);
	dim3 n_blocks ((sz[0] + n_th_x - 1) / n_th_x, (sz[1] + n_th_y - 1) / n_th_y, (sz[2] + n_th_z - 1) / n_th_z);

	precFactorElasticity <<< n_blocks, n_threads >>> (ux_hat, uy_hat, uz_hat, fx_hat, fy_hat, fz_hat, lam_avg, mu_avg, screen_avg);

	hipDeviceSynchronize();
	cudaCheckKernelError ();
}
  
void computeStressQuantsCuda(ScalarType **gradu_ptr, ScalarType *jac_ptr, ScalarType *trace_ptr, ScalarType *max_shear_ptr, ScalarType *mu_ptr, ScalarType *lam_ptr, int64_t sz) {
	int n_th = N_THREADS;
	
	computeStressQuants <<< (sz + n_th - 1) / n_th, n_th >>> (gradu_ptr, jac_ptr, trace_ptr, max_shear_ptr, mu_ptr, lam_ptr, sz);

	hipDeviceSynchronize();
	cudaCheckKernelError ();
}
